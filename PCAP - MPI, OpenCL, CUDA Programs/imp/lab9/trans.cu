
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void transpose(int *a,int *b,int m){
	int rows = threadIdx.y; 
	int cols = threadIdx.x;

	//b[cols*(*m)+rows]=a[cols*(*m)+rows];
	b[cols*(m)+rows]=a[rows*(m)+cols];
}
int main(){
	int m;
	printf("Enter m");
	scanf("%d",&m);
	int a[m*m],b[m*m];
	printf("Enter elements \n");
	for(int i = 0;i<m*m;i++)
		scanf("%d",&a[i]);
	int *d_a,*d_b;
	int d_m;
	hipMalloc((void**)&d_a,m*m*sizeof(int));
	hipMalloc((void**)&d_b,m*m*sizeof(int));
	hipMalloc((void**)&d_m,sizeof(int));
	hipMemcpy(d_a,&a,m*m*sizeof(int),hipMemcpyHostToDevice);
	dim3 griddim(1,1,1);
	dim3 blockdim(m,m,1);
	transpose<<<griddim,blockdim>>>(d_a,d_b,m);
	hipMemcpy(&b,d_b,m*m*sizeof(int),hipMemcpyDeviceToHost);
	printf("Transpose of the matrix is \n");
	for(int i = 0;i<m*m;i++)
		printf("%d ",b[i]);

	hipFree(d_a);
	hipFree(d_b);
	//cudaFree(d_m);
	return 0;
}
