
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mul1 (int *A, int *B, int *C, int height, int width) {
    int idx = blockIdx.x;
    int sum=0;
    for(int i=0; i<width; i++){
        sum=0;
        for(int j=0; j<width; j++)
            sum+=A[idx*width+j]*B[j*width+i];
        C[idx*width+i]=sum;
    }
}

__global__ void mul2 (int *A, int *B, int *C, int height, int width) {
    int idx = blockIdx.x;
    int sum=0;
    for(int i=0; i<height; i++){
        sum=0;
        for(int j=0; j<width; j++)
            sum+=A[idx*width+j]*B[j*width+i];
        C[idx*width+i]=sum;
    }
}

__global__ void mul3 (int *A, int *B, int *C, int height, int width) {
    int rid = blockIdx.y;
    int cid = threadIdx.x;
    int sum=0;
    for(int i=0; i<width; i++){
            sum+=A[rid*width+i]*B[i*width+cid];
    }
        C[rid*width+cid]=sum;
}


int main() {

    int N, M;
    printf("\nEnter The Value Of N and M: ");
    scanf("%d %d",&N,&M); //row

    int row=N, col=M;

    int sizeMatrix=N*M;
    // Host copies of the variables
    int A[sizeMatrix], B[sizeMatrix], C[sizeMatrix];

    int i, j;
    for(i=0; i<sizeMatrix; i++)
        scanf("%d",&A[i]);
    for(j=0; j<sizeMatrix; j++)
        scanf("%d",&B[j]);

    // Device copies of the variables
    int *d_a, *d_b, *d_c;

    int size = sizeof(int) * sizeMatrix;

    // Allocate Memories to device copies of the objects
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &B, size, hipMemcpyHostToDevice);

    // Launch kernel onto the device
    mul1<<<row, 1>>>(d_a, d_b, d_c,row,col);

    //mul2<<<row, 1>>>(d_a, d_b, d_c,row,col);

    //dim3 dimBlock = (row,col);
    //mul3<<<1, dimBlock>>>(d_a, d_b, d_c,row,col);

    // Copy the result back to the host
    hipMemcpy(&C, d_c, size, hipMemcpyDeviceToHost);

    printf("A:\n");
    for (j = 0; j < sizeMatrix; ++j) {
        printf("%d\t", A[j]);
    }
    printf("\n");

    printf("B:\n");
    for (j = 0; j < sizeMatrix; ++j) {
        printf("%d\t", B[j]);
    }
    printf("\n");

    printf("A * B:\n");
    for (j = 0; j < sizeMatrix; ++j) {
        printf("%d\t", C[j]);
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    getchar();

    return 0;
}