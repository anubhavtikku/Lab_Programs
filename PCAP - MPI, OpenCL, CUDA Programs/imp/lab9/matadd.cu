
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void add (int *A, int *B, int *C,int M,int N) {
    
    //Row Wise
    int idx = threadIdx.x;
    for (int i = 0; i < M; ++i) 
        C[i + M * idx] = A[i + M * idx] + B[i + M * idx];
    
    //Column Wise
    //int idx = threadIdx.x;
    //for (int i = 0; i < N; ++i, idx += M) 
        //C[idx] = A[idx] + B[idx];

    //Element Wise
    //int idx = blockDim.x * blockIdx.x + threadIdx.x;
    //C[idx] = A[idx] + B[idx];


}

int main () {
    int M,N;
    printf("N rows and M columns \n");
    scanf("%d%d",&N,&M);
    int A[M * N], B[M * N], C[M * N];
    int i, j;
    printf("A ELEMENTS \n");
    for (i = 0; i < M * N; ++i) {
        scanf("%d",&A[i]);
    }
    printf("B ELEMENTS \n");
    for (i = 0; i < M * N; ++i) {
        scanf("%d",&B[i]);
    }

    int *d_a, *d_b, *d_c;

    int size = sizeof(int) * M * N;

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);


    hipMemcpy(d_a, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &B, size, hipMemcpyHostToDevice);
    //Row and Column both
    add<<<1, N>>>(d_a, d_b, d_c,M,N);
    //Element Wise
    //add<<<M, N>>>(d_a, d_b, d_c,M,N);
    hipMemcpy(&C, d_c, size, hipMemcpyDeviceToHost);

    printf("A:\n");
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            printf("%d\t", A[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    printf("B:\n");
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            printf("%d\t", B[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    printf("A + B:\n");
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            printf("%d\t", C[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    getchar();

    return 0;
}