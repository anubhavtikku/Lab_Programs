#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>



__device__ int getGlobalIdx_1D_1D(){
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void find(char* A, char* B, int* N, int* C, int len){
	int id=getGlobalIdx_1D_1D();
	int j;
	int li = N[id]+1;
	int ri = N[id+1];
	int flag=1;
	if(ri-li!=len){
		C[id]=0;
	}else{
	for(int j=li;j<ri;j++){
		if(B[j-li]!=A[j]){
			flag=1;
		}
	}
	C[id] = flag;
	}
}


int main(){
	char A[1000];
	char B[1000];
	int C[1000]={0};
	char ca;
	gets(A);
	scanf("%c",&ca);
	gets(B);
	int len1 = strlen(A);
	int len2 = strlen(B);
	int count=0;
	for(int i=0; i<strlen(A); i++){
		if(A[i]==' '){
			count = count+1;
		}
	}
	count = count+2;
	int N[count];
	int same[count-1];
	N[0] = -1;
	N[count-1] = strlen(A);
	int j=1;
	for(int i=0; i<strlen(A); i++)
		if(A[i]==' ')
			N[j++] = i;
	char (*pA), (*pB);
	int (*pC),(*pN);

	hipMalloc((void**)&pA, (len1+1)*sizeof(char));
	hipMalloc((void**)&pB, (len2+1)*sizeof(char));
	hipMalloc((void**)&pN, (count)*sizeof(int));
	hipMalloc((void**)&pC, (count-1)*sizeof(int));

	hipMemcpy(pA, A, (len1+1)*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(pB, B, (len2+1)*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(pN, N, (count)*sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(pC, C, (len1-len2+1)*sizeof(int), hipMemcpyHostToDevice);

	int numBlocks = 1;
	int threadsPerBlock = count-1;
	find<<<numBlocks,threadsPerBlock>>>(pA,pB,pN,pC,len2);

	hipMemcpy(C, pC, (count-1)*sizeof(int), hipMemcpyDeviceToHost);

	int i; printf("C = \n");
	for(i=0;i<count-1;i++){
	    printf("%d ", C[i]);
	}
	printf("\n%d\n",len2);


	hipFree(pA);
	hipFree(pB);
	hipFree(pC);

	printf("\n");

	return 0;

}
