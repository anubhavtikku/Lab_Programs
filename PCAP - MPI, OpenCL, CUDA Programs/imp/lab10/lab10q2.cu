#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N  9

__device__ int getGlobalIdx_1D_1D(){
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void find( char* A,  char* B,int len){
	int id=getGlobalIdx_1D_1D();
	B[id] = A[len-id-1];
}


int main(){
	char A[1000]={'\0'};
	char B[1000]={'\0'};
	char ca;
	gets(A);
	scanf("%c",&ca);
	int len1 = strlen(A);

	char (*pA), (*pB);

	hipMalloc((void**)&pA, (len1)*sizeof(char));
	hipMalloc((void**)&pB, (len1)*sizeof(char));

	hipMemcpy(pA, A, (len1)*sizeof(char), hipMemcpyHostToDevice);
	//hipMemcpy(pC, C, (len1-len2+1)*sizeof(int), hipMemcpyHostToDevice);

	int numBlocks = 1;
	int threadsPerBlock = len1;
	find<<<numBlocks,threadsPerBlock>>>(pA,pB,len1);

	hipMemcpy(B, pB, (len1)*sizeof(char), hipMemcpyDeviceToHost);

	printf("%s\n",B);


	hipFree(pA);
	hipFree(pB);

	printf("\n");

	return 0;

}
