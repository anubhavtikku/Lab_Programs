
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
__global__ void substring(char *a,char *b,int *m,int *n,int *match){
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
	
	if(idx<=((*m)-(*n)))
	{
		int flag = 1;
		for(int i = 0;i<(*n);i++){
			if(a[idx+i]!=b[i]){
				flag=0;
				break;
			}
		}
	match[idx]=flag;		
	
	}
	


}
int main(){
	int m,n,pos=-1;
	char s[100],p[100];
	printf("Enter string \n");
	scanf("%s",s);
	m=strlen(s);
	printf("Enter pattern \n");
	scanf("%s",p);
	n=strlen(p);
	int match[m-n+1];
	for(int i =0;i<m-n;i++)
		match[i]=2;
	char *d_s,*d_p;
	int *d_m,*d_n,*d_match;
	hipMalloc((void**)&d_s,m*sizeof(char));
	hipMalloc((void**)&d_p,n*sizeof(char));
	hipMalloc((void**)&d_m,sizeof(int));
	hipMalloc((void**)&d_n,sizeof(int));
	hipMalloc((void**)&d_match,(m-n+1)*sizeof(int));
	hipMemcpy(d_s,s,m*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_p,p,n*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_m,&m,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_n,&n,sizeof(int),hipMemcpyHostToDevice);
	dim3 griddim(1,1,1);
	dim3 blockdim(m-n+1,1,1);
	substring<<<griddim,blockdim>>>(d_s,d_p,d_m,d_n,d_match);
	hipMemcpy(match,d_match,(m-n+1)*sizeof(int),hipMemcpyDeviceToHost);
	for(int i =0;i<=(m-n);i++){
		if(match[i]==1){
			pos=i;
		}
	}
	printf("Substring found at position %d",pos);

	hipFree(d_s);
	hipFree(d_p);
	hipFree(d_m);
	hipFree(d_n);
	hipFree(d_match);
	return 0;
}
