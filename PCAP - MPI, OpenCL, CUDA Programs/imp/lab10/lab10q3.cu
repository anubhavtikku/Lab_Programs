#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N  9

__device__ int getGlobalIdx_1D_1D(){
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void find( char* A,  int* B){
	int id=getGlobalIdx_1D_1D();
	int n = A[id];
	int i = 1;
	int bin = 0;
	while(n>0){
		int rem = n%2;
		bin = bin + rem*i;
		i=i*10;
		n=n/2;
	}
	B[id] = bin;
}


int main(){
	char A[1000]={'\0'};
	int B[1000]={0};
	char ca;
	gets(A);
	scanf("%c",&ca);
	int len1 = strlen(A);

	char (*pA);
	int (*pB);

	hipMalloc((void**)&pA, (len1)*sizeof(char));
	hipMalloc((void**)&pB, (len1)*sizeof(int));

	hipMemcpy(pA, A, (len1)*sizeof(char), hipMemcpyHostToDevice);
	//hipMemcpy(pC, C, (len1-len2+1)*sizeof(int), hipMemcpyHostToDevice);

	int numBlocks = 1;
	int threadsPerBlock = len1;
	find<<<numBlocks,threadsPerBlock>>>(pA,pB);

	hipMemcpy(B, pB, (len1)*sizeof(int), hipMemcpyDeviceToHost);

	int i; printf("C = \n");
	for(i=0;i<len1;i++){
	    printf("%d ", B[i]);
	}
	printf("\n");


	hipFree(pA);
	hipFree(pB);

	printf("\n");

	return 0;

}
