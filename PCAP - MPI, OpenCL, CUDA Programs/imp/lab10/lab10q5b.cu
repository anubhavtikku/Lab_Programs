#include "hip/hip_runtime.h"

#include<stdio.h>

__device__ int getGlobalIdx_2D_2D(){
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}
__global__ void add_matrix(int* a, int* b, int* c,int colsize1,int colsize2)
{
	int id = getGlobalIdx_2D_2D();
	c[id] = 0;
	for(int i=0;i<colsize1;i++)
	{
	c[id] = c[id]+a[((blockIdx.y*blockDim.y)+threadIdx.y)* (colsize1) + i]*b[i*(colsize2) + ((blockIdx.x*blockDim.x)+threadIdx.x)];
	}
}


int main()
{
	int row1, col1, row2, col2;
	printf("enter row1 & col1\n");
	scanf("%d",&row1);
	scanf("%d",&col1);
	printf("enter row2 & col2\n");

	scanf("%d",&row2);
	scanf("%d",&col2);

	int a[1000]={0}, b[1000]={0}, c[1000]={0};

	if(col1!=row2)
		exit(0);

	int *d_a, *d_b, *d_c, *col_size1;

	int size1 = sizeof(int)*row1*col1;
	int size2 = sizeof(int)*row2*col2;
	int size3 = sizeof(int)*1000;
	int size_int = sizeof(int);

	hipMalloc((void**)&d_a,size1);
	hipMalloc((void**)&d_b,size2);
	hipMalloc((void**)&d_c,size3);
	hipMalloc((void**)&col_size1, size_int);


	for(int i = 0;i<row1*col1;i++)
	{
		a[i] = 1;

	}
	for(int j=0;j<row2*col2;j++)
	{
		b[j] = 1;
	}

	if(col2%2==0){
		col2 = col2+1;
		for(int j=row2*(col2-1);j<row2*col2;j++){
			b[j]=0;
		}
	}
	if(row1%2==0){
		row1 = row1+1;
		for(int j=(row1-1)*(col1);j<row1*col1;j++){
			a[j]=0;
		}
	}
	if(col1%2==0){
			col1 = col1+1;
			for(int j=row1*(col1-1);j<row1*col1;j++){
				a[j]=0;
			}
		}
		if(row2%2==0){
			row2 = row2+1;
			for(int j=(row2-1)*(col2);j<row2*col2;j++){
				b[j]=0;
			}
		}
	hipMemcpy(d_a, &a, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size2, hipMemcpyHostToDevice);
	hipMemcpy(col_size1, &col1, size_int, hipMemcpyHostToDevice);
	int WC = col2;
	int HC = row1;
	dim3 numBlocks(WC%2==0?(WC/2):((WC/2)+1),HC%2==0?(HC/2):((HC/2)+1));
	dim3 threadsPerBlock(2,2);
	add_matrix<<<threadsPerBlock,numBlocks>>>(d_a, d_b, d_c, col1,col2);

	hipMemcpy(&c, d_c, size3, hipMemcpyDeviceToHost);
	for(int i=0;i<row1*col2;i++)
	{
		if(i%col2 == 0 && i!=0)
			printf("\n");
		printf("%d ", c[i]);

	}
	printf("\n");
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}
