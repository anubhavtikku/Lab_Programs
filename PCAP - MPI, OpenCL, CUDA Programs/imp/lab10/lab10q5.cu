#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define WA  2
#define HA  9
#define WB  2
#define HB  9
#define WC  2
#define HC  9


__device__ int getGlobalIdx_2D_2D(){
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__global__ void addMat( int* A,  int* B, int* C){
	int id = getGlobalIdx_2D_2D();
	if(id<gridDim.x*gridDim.y*blockDim.x*blockDim.y){
		C[id] = A[id]+B[id];
	}
}

int main(){

int A[WA*HA];
int B[WB*HB];
int C[WC*HC]={0};
for(int i=0;i<WA*HA;i++)
		A[i]=1;
for(int i=0;i<WB*HB;i++)
		B[i]=1;
int (*pA), (*pB), (*pC);

hipMalloc((void**)&pA, (WA*HA)*sizeof(int));
hipMalloc((void**)&pB, (WB*HB)*sizeof(int));
hipMalloc((void**)&pC, (WC*HC)*sizeof(int));

hipMemcpy(pA, A, (WA*HA)*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(pB, B, (WB*HB)*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(pC, C, (WC*HC)*sizeof(int), hipMemcpyHostToDevice);

dim3 numBlocks(HC%2==0?(HC/2):(HC/2+1),WC%2==0?(WC/2):(WC/2+1));
dim3 threadsPerBlock(2,2);
addMat<<<numBlocks,threadsPerBlock>>>(pA,pB,pC);

hipMemcpy(C, pC, (WC*HC)*sizeof(int), hipMemcpyDeviceToHost);

int i, j=0;; printf("C = \n");
for(i=0;i<WC*HC;i++){
	if(j%WC==0) printf(" \n");
    printf("%d ", C[i]);
	j++;
    }
    printf("\n");


hipFree(pA);
hipFree(pB);
hipFree(pC);

printf("\n");

return 0;
}
