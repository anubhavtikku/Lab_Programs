#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N  9

__device__ int getGlobalIdx_1D_1D(){
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void find( char* A,  char* B, int* C, int len){
	int id=getGlobalIdx_1D_1D();
	int flag=0;
	for(int i=0;i<len;i++){
		if(A[id+i]!=B[i]){
			flag=1;
		}
	}
	if(flag==0){
		C[id]=1;
	}else{
		C[id]=0;
	}
}


int main(){
	char A[1000];
	char B[1000];
	int C[1000]={0};
	char ca;
	gets(A);
	scanf("%c",&ca);
	gets(B);
	int len1 = strlen(A);
	int len2 = strlen(B);

	char (*pA), (*pB);
	int (*pC);

	hipMalloc((void**)&pA, (len1)*sizeof(char));
	hipMalloc((void**)&pB, (len2)*sizeof(char));
	hipMalloc((void**)&pC, (len1-len2+1)*sizeof(int));

	hipMemcpy(pA, A, (len1)*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(pB, B, (len2)*sizeof(char), hipMemcpyHostToDevice);
	//hipMemcpy(pC, C, (len1-len2+1)*sizeof(int), hipMemcpyHostToDevice);

	int numBlocks = 1;
	int threadsPerBlock = len1-len2+1;
	find<<<numBlocks,threadsPerBlock>>>(pA,pB,pC,len2);

	hipMemcpy(C, pC, (len1-len2+1)*sizeof(int), hipMemcpyDeviceToHost);

	int i; printf("C = \n");
	for(i=0;i<len1-len2+1;i++){
	    printf("%d ", C[i]);
	}
	printf("\n%d\n",len2);


	hipFree(pA);
	hipFree(pB);
	hipFree(pC);

	printf("\n");

	return 0;

}
