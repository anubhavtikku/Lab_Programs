
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void add(int *a,int *b,int *c){
	//int idx=blockIdx.x;
	int idx=threadIdx.x;
	c[idx]=a[idx]+b[idx];
}

int main()
{
int a[30],b[30],c[30];
int *d_a,*d_b,*d_c;
int i,n;
printf("Enter n ");
scanf("%d",&n);
printf("Enter values of A ");
for(i=0;i<n;i++)
	scanf("%d",&a[i]);
printf("Enter values of B");
for(i=0;i<n;i++)
	scanf("%d",&b[i]);
int size=n*sizeof(int);
hipMalloc((void **)&d_a,size);
hipMalloc((void **)&d_b,size);
hipMalloc((void **)&d_c,size);

hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
//For blocks
//add<<<n,1>>>(d_a,d_b,d_c);
//For threads
add<<<1,n>>>(d_a,d_b,d_c);
hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);
for(i=0;i<n;i++)
	printf("%d\t",c[i]);
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}

